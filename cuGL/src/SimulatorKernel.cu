#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudacheck.h"

__constant__ int ConstMem[2];

__global__ void SimulatorUpdateMagneticFieldsKernel(float* hz, float* ex, float* ey,
    float* Chzh, float* Chzex, float* Chzey)
{
    int indx = threadIdx.x + blockIdx.x * blockDim.x;
    int indy = threadIdx.y + blockIdx.y * blockDim.y;
    
   
    int offset = (indx + 16) + ConstMem[0] * (indy + 16);
    int offsetC = indx + ConstMem[1] * indy;
    int offsetyp = offset + ConstMem[0];
    int offsetxp = offset + 1;

    hz[offset] = Chzh[offsetC] * hz[offset] 
        + Chzex[offsetC] * (ex[offsetyp] - ex[offset]) 
        + Chzey[offsetC] * (ey[offsetxp] - ey[offset]);

}

__global__ void SimulatorUpdateElectricFieldsKernel(float* hz, float* ex, float* ey,
    float* Cexe, float* Cexhz, float* Ceye, float* Ceyhz)
{
    int indx = threadIdx.x + blockIdx.x * blockDim.x;
    int indy = threadIdx.y + blockIdx.y * blockDim.y;
    
    int offset = (indx + 16) + ConstMem[0] * (indy + 16);
    int offsetC = indx + ConstMem[1] * indy;
    int offsetyn = offset - ConstMem[0];
    int offsetxn = offset - 1;


   
    ex[offset] = Cexe[offsetC] * ex[offset]
        + Cexhz[offsetC] * (hz[offset] - hz[offsetyn]);
    ey[offset] = Ceye[offsetC] * ey[offset]
        + Ceyhz[offsetC] * (hz[offset] - hz[offsetxn]);

}

__global__ void Check(float* data, int Mconst)
{
    int indx = threadIdx.x + blockIdx.x * blockDim.x;
    int indy = threadIdx.y + blockIdx.y * blockDim.y;
    int offsetC = indx + Mconst * indy;
    if (data[offsetC] != 1.0f)
        printf("Error! ThreadId is %d , %d\n value is %f", indx, indy, data[offsetC]);
}


void SimulatorUpdateMagneticFieldsInterface(float* Hz, float* Ex, float* Ey,
    float* Chzh, float* Chzex, float* Chzey,
	dim3 Grids, dim3 Threads)
{

    SimulatorUpdateMagneticFieldsKernel << <Grids, Threads >> >
		(Hz, Ex, Ey, Chzh, Chzex, Chzey);
}

void SimulatorUpdateElectricFieldsInterface(float* Hz, float* Ex, float* Ey,
    float* Cexe, float* Cexhz, float* Ceye, float* Ceyhz,
    dim3 Grids, dim3 Threads)
{

    SimulatorUpdateElectricFieldsKernel << <Grids, Threads >> >
        (Hz, Ex, Ey, Cexe, Cexhz, Ceye, Ceyhz);
}


void SimulatorSetConstInterface(int* src, int num)
{
    static bool Done = false;
    if (!Done)
    {
        CUDACheck(hipMemcpyToSymbol(HIP_SYMBOL(ConstMem), src, num * sizeof(int)));
        Done = true;
    }
}


